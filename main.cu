#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <cutf/math.hpp>
#include <cutf/type.hpp>
#include <cutf/memory.hpp>
#include <cutf/cublas.hpp>
#include <cutf/device.hpp>
#include <cutf/error.hpp>

template <class T>
__device__ T sign(const T v){
	if( v < cutf::cuda::type::cast<T>(0.0f) ){
		return -v;
	}else{
		return v;
	}
}

template <class T>
__global__ void kernel_if(const T* const a, T* const b){
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	b[tid] = sign(a[tid]);
}
template <class T>
__global__ void kernel_cutf(const T* const a, T* const b){
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	b[tid] = cutf::cuda::math::sign(a[tid]);
}

template <std::size_t N, std::size_t C, class T, class Func>
void test(Func func){
	std::cout<<__func__<<std::endl;
	auto dF = cutf::cuda::memory::get_device_unique_ptr<T>(N);
	auto hF = cutf::cuda::memory::get_host_unique_ptr<T>(N);
	auto dI = cutf::cuda::memory::get_device_unique_ptr<T>(N);
	auto hI = cutf::cuda::memory::get_host_unique_ptr<T>(N);
	for(auto i = decltype(N)(0); i < N; i++){
		hF.get()[i] = cutf::cuda::type::cast<T>((static_cast<float>(N/2) - i) * 10.0f);
	}

	cutf::cuda::memory::copy(dF.get(), hF.get(), N);

	for(std::size_t c = 0; c < C; c++)
		func(dF.get(), dI.get());

	cutf::cuda::memory::copy(hI.get(), dI.get(), N);
}

template <std::size_t N, class T>
void test_if(const T* const a, T* const b){
	kernel_if<T><<<N, 1>>>(a, b);
}
template <std::size_t N, class T>
void test_cutf(const T* const a, T* const b){
	kernel_cutf<T><<<N, 1>>>(a, b);
}


int main(){
	constexpr std::size_t N = 1 << 10;
	constexpr std::size_t C = 1 << 13;

	test<N, C, half>(test_if<N, half>);
	test<N, C, float>(test_if<N, float>);
	test<N, C, double>(test_if<N, double>);
	test<N, C, half>(test_cutf<N, half>);
	test<N, C, float>(test_cutf<N, float>);
	test<N, C, double>(test_cutf<N, double>);
}
